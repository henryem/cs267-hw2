#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;
//
//  benchmarking program
//

struct GridMetadata {
  //FIXME
  double grid_square_size;
  int num_grid_squares_per_side;
  int num_grid_squares;

  GridMetadata() { } //FIXME

  GridMetadata(double grid_square_size_v, int num_grid_squares_per_side_v):
    grid_square_size(grid_square_size_v),
    num_grid_squares_per_side(num_grid_squares_per_side_v),
    num_grid_squares(num_grid_squares_per_side_v*num_grid_squares_per_side_v) { }
};

inline int flat_square_idx(const particle_t& p, const GridMetadata& g) {
  return 0; //FIXME
}

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t* particles, int n) {
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++) {
    apply_force_gpu(particles[tid], particles[j]);
  }
}

__global__ void move_gpu(particle_t * particles, int n, double size) {

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
  //
  //  slightly simplified Velocity Verlet integration
  //  conserves energy better than explicit Euler method
  //
  p->vx += p->ax * dt;
  p->vy += p->ay * dt;
  p->x  += p->vx * dt;
  p->y  += p->vy * dt;

  //
  //  bounce from walls
  //
  while(p->x < 0 || p->x > size) {
    p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
    p->vx = -(p->vx);
  }
  while(p->y < 0 || p->y > size) {
    p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
    p->vy = -(p->vy);
  }

}



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if (find_option( argc, argv, "-h" ) >= 0) {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    const int n = read_int( argc, argv, "-n", 1000 );
    const bool fast = (find_option( argc, argv, "-no" ) != -1);
    const char *savename = read_string( argc, argv, "-o", NULL );
    const char *sumname = read_string( argc, argv, "-s", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    const double size = set_size( n );

    // Particles are stored in an array of blocks of squares.  Each square
    // is large enough that particles can only move across 1 square per
    // simulated time step, but small enough that the expected number of
    // particles (and hopefully the maximum number) per square is a small
    // constant.
    // The block structure and squares are constant, but the square locations
    // of particles changes across iterations and is not fixed.  We use an
    // expandable Thrust vector for convenience.
    const double grid_square_size = cutoff + 0.0001;
    const int num_grid_squares_per_side = div_round_up_f(size, grid_square_size);
    const int num_grid_squares = num_grid_squares_per_side * num_grid_squares_per_side;
    //FIXME
    const GridMetadata grid(grid_square_size, num_grid_squares_per_side);
    thrust::host_vector<thrust::host_vector<particle_t> > ps(num_grid_squares);

    particle_t* particles = init_particles(n);
    // Generate the grid initially, serially, on the host.  This is slow, but
    // it cannot really be done efficiently on the GPU.  Note that the
    // particles are _copied_ into the grid so that there will be good
    // cache locality in the GPU code; the grid is the authoritative store
    // of the particles.
    for (unsigned int i = 0; i < n; i++) {
      particle_t& p = particles[i];
      ps[flat_square_idx(p, grid)].push_back(p);
    }

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
//    hipMemcpy(d_particles, (*particles).data(), (*particles).size() * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ ) {
      // First, we must build the grid.



      //
      //  compute forces
      //

      int blks = div_round_up(n, NUM_THREADS);
      compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);

      //
      //  move particles
      //
      move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);

      //
      //  save if necessary
      //
      if( fsave && (step%SAVEFREQ) == 0 ) {
        // Copy the particles back to the CPU
        hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
        save( fsave, n, particles);
      }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    hipFree(d_particles);
    free(particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
