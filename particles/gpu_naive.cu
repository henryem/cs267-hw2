#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
    return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, int n)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
  //
  //  slightly simplified Velocity Verlet integration
  //  conserves energy better than explicit Euler method
  //
  p->vx += p->ax * dt;
  p->vy += p->ay * dt;
  p->x  += p->vx * dt;
  p->y  += p->vy * dt;

  //
  //  bounce from walls
  //
  while( p->x < 0 || p->x > size )
  {
    p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
    p->vx = -(p->vx);
  }
  while( p->y < 0 || p->y > size )
  {
    p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
    p->vy = -(p->vy);
  }

}



int main( int argc, char **argv )
{    
  // This takes a few seconds to initialize the runtime
  hipDeviceSynchronize();

  if( find_option( argc, argv, "-h" ) >= 0 )
  {
    printf( "Options:\n" );
    printf( "-h to see this help\n" );
    printf( "-n <int> to set the number of particles\n" );
    printf( "-o <filename> to specify the output file name\n" );
    return 0;
  }

  int n = read_int( argc, argv, "-n", 1000 );

  char *savename = read_string( argc, argv, "-o", NULL );

  FILE *fsave = savename ? fopen( savename, "w" ) : NULL;

  const double size = set_size( n );
  particle_t *particles = init_particles(n);

  // GPU particle data structure
  particle_t * d_particles;
  hipMalloc((void **) &d_particles, n * sizeof(particle_t));

  hipDeviceSynchronize();
  double copy_time = read_timer( );

  // Copy the particles to the GPU
  hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  copy_time = read_timer( ) - copy_time;

  //
  //  simulate a number of time steps
  //
  hipDeviceSynchronize();
  double simulation_time = read_timer( );

  for( int step = 0; step < NSTEPS; step++ )
  {
    //
    //  compute forces
    //

    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);

    //
    //  move particles
    //
    move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);

    //
    //  save if necessary
    //
    if( fsave && (step%SAVEFREQ) == 0 ) {
      // Copy the particles back to the CPU
      hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
      save( fsave, n, particles);
    }
  }
  hipDeviceSynchronize();
  simulation_time = read_timer( ) - simulation_time;

  printf( "CPU-GPU copy time = %g seconds\n", copy_time);
  printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

  free( particles );
  hipFree(d_particles);
  if( fsave )
    fclose( fsave );

  return 0;
}
