#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/for_each.h>
#include <thrust/scatter.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include "common.h"

#define NUM_THREADS 256

struct GridMetadata {
  int num_particles;
  // Approximately, but not exactly, equal to square_size*side_count.
  double side_size;
  double square_size;
  int side_count;
  int count;

  GridMetadata(int num_particles_v, double side_size_v, double square_size_v, int side_count_v):
    num_particles(num_particles_v),
    side_size(side_size_v),
    square_size(square_size_v),
    side_count(side_count_v),
    count(side_count_v*side_count_v) { }

  __host__ __device__ int particle_to_flat_square_idx(const particle_t& p) const {
    int x_idx = (int) (p.x / square_size);
    int y_idx = (int) (p.y / square_size);
    return square_to_flat_square_idx(x_idx, y_idx);
  }

  __host__ __device__ int square_to_flat_square_idx(int square_x, int square_y) const {
    const int idx = square_x + side_count*square_y;
    return idx;
  }
};

// Functor to map a particle to its grid location.
struct GridSquareCmIndex : public thrust::unary_function<particle_t&, int> {
  const GridMetadata g;

  GridSquareCmIndex(const GridMetadata g_v) : g(g_v) { }

  __host__ __device__ int operator()(particle_t& p) const {
    return g.particle_to_flat_square_idx(p);
  }
};

// Functor to move a particle.
struct MoveParticle : public thrust::unary_function<particle_t&, void> {
  const GridMetadata g;

  MoveParticle(const GridMetadata g_v) : g(g_v) { }

  __host__ __device__ void operator()(particle_t& p) {
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p.vx += p.ax * dt;
    p.vy += p.ay * dt;
    p.x  += p.vx * dt;
    p.y  += p.vy * dt;

    //
    //  bounce from walls
    //
    while(p.x < 0 || p.x > g.side_size) {
      p.x  = p.x < 0 ? -(p.x) : 2*g.side_size-p.x;
      p.vx = -(p.vx);
    }
    while(p.y < 0 || p.y > g.side_size) {
      p.y  = p.y < 0 ? -(p.y) : 2*g.side_size-p.y;
      p.vy = -(p.vy);
    }
  }
};

struct IgnoreZeroPredicate : public thrust::unary_function<int, bool> {
  __host__ __device__ bool operator()(int i) {
    return i != 0;
  }
};

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__device__ int num_particles_in_square(int square_idx, const int* grid_offsets, const GridMetadata grid) {
  int square_start = grid_offsets[square_idx];
  // For all but the last grid square, the number of particles in the square
  // equals the next square's offset minus its own offset.  The last square
  // just has all the remaining particles.
  int square_end = (square_idx < (grid.count - 1)) ? grid_offsets[square_idx+1] : grid.num_particles;
  return square_end - square_start;
}

__global__ void compute_forces_gpu (particle_t* particles, const int* grid_offsets, const GridMetadata grid) {
  // Get grid square ID.  Each call to this function computes forces
  // for all of the particles in one grid square.
  const int square_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (square_idx >= grid.count) return;

  const int square_x = square_idx % grid.side_count;
  const int square_y = square_idx / grid.side_count;
  const int first_particle_idx = grid_offsets[square_idx];
  const int num_ps = num_particles_in_square(square_idx, grid_offsets, grid);
  for (int particle_idx = first_particle_idx; particle_idx < first_particle_idx + num_ps; particle_idx++) {
    particles[particle_idx].ax = particles[particle_idx].ay = 0;
    // Iterate only over neighboring grid squares.  Note that we could reorder
    // this to access each neighbor only once, which would probably be better.
    for (int y_offset = -1; y_offset <= 1; y_offset++) {
      const int neighbor_y = square_y + y_offset;
      if (neighbor_y < 0 || neighbor_y >= grid.side_count) {
        continue;
      }
      for (int x_offset = -1; x_offset <= 1; x_offset++) {
        const int neighbor_x = square_x + x_offset;
        if (neighbor_x < 0 || neighbor_x >= grid.side_count) {
          continue;
        }
        const int neighbor_square_idx = grid.square_to_flat_square_idx(neighbor_x, neighbor_y);
        // Now we iterate over all the particles in the neighbor and apply
        // forces to our particle.
        const int first_neighbor_particle_idx = grid_offsets[neighbor_square_idx];
        const int num_neighbor_ps = num_particles_in_square(neighbor_square_idx, grid_offsets, grid);
        for (int neighbor_particle_idx = first_neighbor_particle_idx;
            neighbor_particle_idx < first_neighbor_particle_idx + num_neighbor_ps;
            neighbor_particle_idx++) {
          apply_force_gpu(particles[particle_idx], particles[neighbor_particle_idx]);
        }
      }
    }
  }
}

/*
 * @param particle_square_idx_storage is a preallocated device array of size
 *   at least grid.num_particles.  Its value can be arbitrary.  It will be
 *   clobbered.
 * @param grid_offsets will be populated with offsets corresponding to grid
 *  squares.  [grid_offsets[i], grid_offsets[i+1]) (note the inclusiveness!)
 *  is the set of indices in particles for grid square i, after this function
 *  returns.
 * @param grid_idx_storage is a preallocated device array of size at least
 *   grid.count.  Its value can be arbitrary.  It will be clobbered.
 * @param grid_count_storage is a preallocated device array of size at least
 *   grid.count.  Its value can be arbitrary.  It will be clobbered.
 */
void sort_to_bins(
    thrust::device_vector<particle_t>& particles,
    thrust::device_vector<int>& particle_square_idx_storage,
    thrust::device_vector<int>& grid_offsets,
    thrust::device_vector<int>& grid_idx_storage,
    thrust::device_vector<int>& grid_count_storage,
    const GridMetadata& grid) {
  thrust::fill(particle_square_idx_storage.begin(), particle_square_idx_storage.end(), 0);
  thrust::transform(
      particles.begin(),
      particles.end(),
      particle_square_idx_storage.begin(),
      GridSquareCmIndex(grid));

  // Sort the particles by column-major order in the grid.  Thrust offers no
  // sort function that leaves the keys in place, so we must actually allocate
  // memory for the square indices.  Another option is to use a comparison
  // functor; that would avoid memory allocation but end up invoking a less
  // efficient sorting algorithm that works on user-defined functors (rather
  // than the fast int-sorter).  It could be worth trying that, too.
  //
  // I think we could do much better than sort_by_key here.  This makes the
  // algorithm O(n log n) when we can do O(n).  Here is a sketch of a better
  // algorithm that seems to equal the asymptotic efficiency of a graph-based
  // or message-passing method, while having very low constant factors when
  // implemented on the GPU:
  //  * Associate with each particle its grid square coordinates (x,y).
  //  * Use a LSH algorithm to hash the grid square coordinates.  LSH means
  //    that the absolute difference between the hashes of two points is (with
  //    high probability) close to the L2 distance of the two points.
  //  * Sort the particles according to the hash.  The sorting algorithm we use
  //    is important:
  //    - Notice that, since particles may move
  //      only a bounded L2 distance in an iteration, there is a bound on the
  //      distance any particle needs to travel in this sorting step.  That is,
  //      the particles are already "almost sorted."
  //    - Therefore we want a sorting algorithm that can be implemented quickly
  //      on the GPU and that takes only O(k N) time, where k is the maximum
  //      (or average) distance moved by any element.
  //    - There are fast algorithms for nearly-sorted lists, but I haven't
  //      found on that is parallelizable yet.  That seems very possible,
  //      though.  So I'm leaving this for future work, due to time
  //      constraints.

  // Unrelated note: Thrust's sort_by_key seems to have a bug (or exercises a
  // memory leak in my code?) that kicks in for particle counts between 18,000
  // and 26,000 and causes this line to crash with a bad_alloc error.  For
  // larger or smaller particle counts, everything seems fine, which is
  // terrifying.
  thrust::sort_by_key(
      particle_square_idx_storage.begin(),
      particle_square_idx_storage.end(),
      particles.begin());

  // Compute the starting offset of each grid square (that is, the index of the
  // first particle in particles contained in each square).
  // I cannot find a totally natural way to do this using Thrust, so we have to
  // jump through some hoops.  We reduce_by_key to compute the number of
  // particles in each nonempty grid square and the corresponding square
  // indices.  Then we scatter into a vector that is initially filled with 0s,
  // mapping each count according to its index.  Then we scan with plus, so
  // that the value at the ith location is the number of particles in grid
  // squares preceding the ith grid square.
  thrust::fill(grid_idx_storage.begin(), grid_count_storage.end(), 0);
  thrust::fill(grid_count_storage.begin(), grid_count_storage.end(), 0);
  thrust::reduce_by_key(
      particle_square_idx_storage.begin(),
      particle_square_idx_storage.end(),
      thrust::make_constant_iterator(1),
      grid_idx_storage.begin(),
      grid_count_storage.begin());
  // Here we set grid_offsets[grid_idx_storage[i]] = grid_count_storage[i]
  // for all i.  Since some grid squares might have been empty, there might
  // be several trailing elements of grid_idx_storage that are zero.  So we
  // use scatter_if() to ignore those.
  thrust::fill(grid_offsets.begin(), grid_offsets.end(), 0);
  thrust::scatter_if(
      grid_count_storage.begin(),
      grid_count_storage.end(),
      grid_idx_storage.begin(),
      grid_count_storage.begin(),
      grid_offsets.begin(),
      IgnoreZeroPredicate());
  thrust::exclusive_scan(
      grid_offsets.begin(),
      grid_offsets.end(),
      grid_offsets.begin());
}

void simulate_forces(thrust::device_vector<particle_t>& particles, const thrust::device_vector<int>& grid_offsets, const GridMetadata& grid) {
  // The communication pattern is not simple, so we have to resort to writing
  // device code ourselves here.
  particle_t* d_particles = thrust::raw_pointer_cast(particles.data());
  const int* d_grid_offsets = thrust::raw_pointer_cast(grid_offsets.data());
  int num_blocks = div_round_up(grid.count, NUM_THREADS);
  compute_forces_gpu<<<num_blocks, NUM_THREADS>>>(d_particles, d_grid_offsets, grid);
}

void simulate_movement(thrust::device_vector<particle_t>& particles, const GridMetadata& grid) {
  thrust::for_each(particles.begin(), particles.end(), MoveParticle(grid));
}



int main( int argc, char **argv )
{    
  // This takes a few seconds to initialize the runtime
  hipDeviceSynchronize();

  if (find_option( argc, argv, "-h" ) >= 0) {
    printf( "Options:\n" );
    printf( "-h to see this help\n" );
    printf( "-n <int> to set the number of particles\n" );
    printf( "-o <filename> to specify the output file name\n" );
    printf( "-no turns off all correctness checks and particle output\n");
    return 0;
  }

  const int n = read_int( argc, argv, "-n", 1000 );
  const bool fast = (find_option( argc, argv, "-no" ) != -1);
  const char *savename = read_string( argc, argv, "-o", NULL );
  const char *sumname = read_string( argc, argv, "-s", NULL );

  FILE *fsave = ((!fast) && savename) ? fopen( savename, "w" ) : NULL;
  const double size = set_size( n );

  // Particles are stored in a flattened array of squares.  Each square
  // is large enough that particles can only move across 1 square per
  // simulated time step, but small enough that the expected number of
  // particles (and hopefully the maximum number) per square is a small
  // constant.
  // Following NVIDIA's example particle simulator, we store the particles
  // in a flattened array and sort them when we need to change the grid
  // structure.  That is, we use one vector
  // of size n to store the particles in sorted order (column major by grid
  // square, with arbitrary order within each square), and a second vector
  // to store the starting offsets for the
  // particles contained in each square.  Then rebuilding the grid involves
  // an in-place sort and recomputing the index.
  const double square_size = cutoff + 0.0001;
  const int side_count = div_round_up_f(size, square_size);
  const GridMetadata grid(n, size, square_size, side_count);

  double init_on_host_time = read_timer();
  //TODO: This part of the initialization is serial.  It is embarrassingly
  // parallel and could easily be done on the GPU.
  particle_t* particles = init_particles(n);
  thrust::host_vector<particle_t> ps(particles, particles+n);
  init_on_host_time = read_timer() - init_on_host_time;

  double init_on_device_time = read_timer();
  // Copy the particles to the GPU.
  thrust::device_vector<particle_t> d_ps = ps;
// Allocate the structure that maps grid locations to offsets in
  // d_ps.  Like d_ps, this will be populated inside the simulation loop,
  // and for now is uninitialized.
  thrust::device_vector<int> d_grid_offsets(grid.count);
  // Allocate scratch space that the algorithm will need.
  thrust::device_vector<int> d_particle_square_idx_storage(grid.num_particles);
  thrust::device_vector<int> d_grid_idx_storage(grid.count);
  thrust::device_vector<int> d_grid_count_storage(grid.count);
  init_on_device_time = read_timer() - init_on_device_time;

  //
  //  simulate a number of time steps
  //
  double simulation_time = read_timer();

  for (int step = 0; step < NSTEPS; step++) {
    // First, we must build the grid.
    sort_to_bins(d_ps, d_particle_square_idx_storage, d_grid_offsets, d_grid_idx_storage, d_grid_count_storage, grid);

    // Now we can simulate forces and movement.
    simulate_forces(d_ps, d_grid_offsets, grid);
    simulate_movement(d_ps, grid);

    if( fsave && (step%SAVEFREQ) == 0 ) {
      // Copy the particles back to the CPU.
      ps = d_ps;
      save( fsave, n, ps.data());
    }
  }
  hipDeviceSynchronize();
  simulation_time = read_timer( ) - simulation_time;

  printf( "CPU-GPU copy time = %g seconds\n", init_on_device_time);
  printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

  free(particles);
  if( fsave )
    fclose( fsave );

  return 0;
}
